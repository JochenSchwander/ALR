#include "hip/hip_runtime.h"
#include "pollard_p1_factorization.h"
#include "math_stuff.h"
#include <math.h>
#include <stdlib.h>

#define DEBUG_POLLARD_P1
#ifdef DEBUG_POLLARD_P1
#include <stdio.h>
#endif // !DEBUG_POLLARD_P1



void pollard_p1_factorization(long long int n, long long int* p, long long int* q, unsigned int *primes, unsigned int primes_length) {
	*p = pollard_p1_factor(n, primes, primes_length);
	*q = n / *p;
}


long long int pollard_p1_factor(long long int n, unsigned int *primes, unsigned int primes_length) {
	long long int b_max = 1000000;
	long long int a_max = 1000;
	long long int b, e, p, i, a, g;

	for (a = 2; a < a_max; a++) {
#ifdef DEBUG_POLLARD_P1
		printf("PP1: a = %lld, ", a);
#endif
		for (b = 2; b < b_max; b++) {
#ifdef DEBUG_POLLARD_P1
			printf("b = %lld, ", b);
#endif
			//calculate e
			e = 1;
			for (i = 0; i < primes_length; i++) {
				p = (long long int) primes[i];
				if (b >= p) {
					e *= power_mod(p, log((long double)b) / log((long double) p), n);
				} else {
					break;
				}
			}
#ifdef DEBUG_POLLARD_P1
			printf("e = %lld, ", e);
#endif
			//check if g is a factor of n
			g = euclidean_gcd(power_mod(a, e - 1, n), n);
#ifdef DEBUG_POLLARD_P1
			printf("g = %lld\n", g);
#endif
			if (g > 1) {
				if (g == n) {
					//found trivial factor n of n
					break;
				} else {
					//found a real factor of n
					return g;
				}
			}
		}
	}

	return n;
}
