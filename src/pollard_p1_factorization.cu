#include "hip/hip_runtime.h"
#include "pollard_p1_factorization.h"
#include "math_stuff.h"
#include <math.h>
#include <stdlib.h>

//#define DEBUG_POLLARD_P1
#ifdef DEBUG_POLLARD_P1
#include <stdio.h>
#endif

//#define POLLARD_P1_V2
#define POLLARD_P1_V1



void pollard_p1_factorization(long long int n, long long int* p, long long int* q, unsigned long int *primes, unsigned long int primes_length) {
	*p = pollard_p1_factor(n, primes, primes_length);
	*q = n / *p;
}


long long int pollard_p1_factor(long long int n, unsigned long int *primes, unsigned long int primes_length) {

	long long int b_max = 1000000;
	long long int a_max = 1000;
	long long int b, e, p, i, a, g;

	for (a = 2; a < a_max; a++) {
#ifdef DEBUG_POLLARD_P1
		printf("PP1: a = %lld\n", a);
#endif
		for (b = 2; b < b_max; b++) {
#ifdef DEBUG_POLLARD_P1
			printf("PP1: b = %lld, ", b);
#endif
			//calculate e
#ifdef POLLARD_P1_V2
			e = 1;
#endif
#ifdef POLLARD_P1_V1
			e = a;
#endif
			for (i = 0; i < primes_length; i++) {
				p = (long long int) primes[i];
				if (b >= p) {
#ifdef POLLARD_P1_V2
					e *= power_mod(p, log((long double)b) / log((long double) p), n);
#endif
#ifdef POLLARD_P1_V1
					e = power_mod(e, p, n);
#endif
				} else {
					break;
				}
			}
#ifdef DEBUG_POLLARD_P1
			printf("e = %lld, ", e);
#endif
			//check if g is a factor of n
#ifdef POLLARD_P1_V2
			g = euclidean_gcd(power_mod(a, e - 1, n), n);
#endif
#ifdef POLLARD_P1_V1
			g = euclidean_gcd(e - 1, n);
#endif
#ifdef DEBUG_POLLARD_P1
			printf("g = %lld\n", g);
#endif
			if (g > 1) {
				if (g == n) {
					//found trivial factor n of n
					break;
				} else {
					//found a real factor of n
					return g;
				}
			}
		}
	}

	return n;
}
