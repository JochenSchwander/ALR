#include "hip/hip_runtime.h"
#include "pollard_p1_factorization.h"
#include "math_stuff.h"
#include <math.h>


void pollard_p1_factorization(long long int n, long long int* p, long long int* q) {
	//TODO primzahlen datei einlesen + als array bereitstellen
	uint16_t *primes;
	uint16_t *primes_length;

	*p = pollard_p1_factor(n, primes, *primes_length);
	*q = n / *p;
}


long long int pollard_p1_factor(long long int n, uint16_t *primes, uint16_t primes_length) {
	long long int b_max = 1000000;
	long long int a_max = 1000;
	long long int b, e, p, i, a, g;

	for (a = 2; a < a_max; a++) {
		for (b = 2; b < b_max; b++) {
			//calculate e
			e = 1;
			for (i = 0; i < primes_length; i++) {
				p = primes[i];
				if (b <= p) {
					e *= power_mod(p, log((long double)b) / log((long double) p), n);
				} else {
					break;
				}
			}

			//check if g is a factor of n
			g = euclidean_gcd(power_mod(a, e - 1, n), n);
			if (g > 1) {
				if (g == n) {
					//found trivial factor n of n
					break;
				} else {
					//found a real factor of n
					return g;
				}
			}
		}
	}

	return n;
}
