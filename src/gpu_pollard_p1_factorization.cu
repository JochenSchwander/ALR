#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "gpu_math_stuff.h"
#include "device.h"
#include <stdbool.h>

#ifdef MACBOOK
int gridSize = 3;
int blockSize = 976;
#endif

#ifdef XMG
int gridSize = 12;
int blockSize = 16;
#endif

//Weichen
//#define DEBUG_GPU_ONLY_CALC

#ifdef DEBUG_GPU_ONLY_CALC
#include <stdio.h>
#include <time.h>
clock_t start, end;
#endif

void setGridSize(int size) {
	gridSize = size;
}

int getGridSize() {
	return gridSize;
}

void setBlockSize(int size) {
	blockSize = size;
}

int getBlockSize() {
	return blockSize;
}

void gpu_pollard_p1_factorization(long long int n, long long int* p, long long int* q, unsigned long int *primes, unsigned long int primes_length) {
	// pointers needed on device
	long long int a, a_max = 1000;
	long long int *n_dev, *p_dev, *a_dev;
	unsigned long int *primes_dev, *primes_length_dev;
	bool factor_not_found = true;
	bool *factor_not_found_dev;


	// allocate memory on device
	hipMalloc((void **) &n_dev, sizeof(long long int));
	hipMalloc((void **) &p_dev, sizeof(long long int));
	hipMalloc((void **) &primes_length_dev, sizeof(unsigned long int));
	hipMalloc((void **) &primes_dev, sizeof(unsigned long int) * primes_length);
	hipMalloc((void **) &a_dev, sizeof(long long int));
	hipMalloc((void **) &factor_not_found_dev, sizeof(bool));

	
	// copy input to device
	hipMemcpy(n_dev, &n, sizeof(long long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_length_dev, &primes_length, sizeof(unsigned long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_dev, primes, sizeof(unsigned long int) * primes_length, hipMemcpyHostToDevice);
	hipMemcpy(factor_not_found_dev, &factor_not_found, sizeof(bool), hipMemcpyHostToDevice);

	for (a = 2; a < a_max && factor_not_found; a++) {
		// copy a in
		hipMemcpy(a_dev, &a, sizeof(long long int), hipMemcpyHostToDevice);

#ifdef DEBUG_GPU_ONLY_CALC
		//measure gpu calculation only
		start = clock();
#endif

		// calculate a prime factor on gpu
		gpu_pollard_p1_factor<<<gridSize,blockSize>>>(n_dev, a_dev, primes_dev, primes_length_dev, p_dev, factor_not_found_dev);
		hipDeviceSynchronize();

#ifdef DEBUG_GPU_ONLY_CALC
		end = clock();
		printf("---> gpu calculation: %lf; a = %lld\n",  (end-start)/(double)CLOCKS_PER_SEC, a);
#endif

		// check if factor allready found
		hipMemcpy(&factor_not_found, factor_not_found_dev, sizeof(bool), hipMemcpyDeviceToHost);
	}

	// copy result to host
	hipMemcpy(p, p_dev, sizeof(long long int), hipMemcpyDeviceToHost);

	// free memory on device
	hipFree(n_dev);
	hipFree(p_dev);
	hipFree(primes_length_dev);
	hipFree(primes_dev);
	hipFree(a_dev);
	hipFree(factor_not_found_dev);

	// calculate other factor on cpu
	*q = n / *p;
}

__global__ void gpu_pollard_p1_factor(long long int *n_in, long long int *a_in, unsigned long int *primes, unsigned long int *primes_length_in, long long int *factor_out, bool *factor_not_found_dev) {
	unsigned int b;
	long long int  e, p, i, g;
	long long int n = *n_in;
	unsigned long int primes_length = *primes_length_in;

	for (b = 2 + blockIdx.x * blockDim.x + threadIdx.x; b < 1000000 && *factor_not_found_dev; b += blockDim.x * gridDim.x) {
		//calculate e
		e = *a_in;
		for (i = 0; i < primes_length; i++) {
			p = (long long int) primes[i];
			if (b >= p) {
				e = gpu_power_mod(e, p, n);
			} else {
				break;
			}
		}

		//check if g is a factor of n
		g = gpu_euclidean_gcd(e - 1, n);
		if (g > 1) {
			if (g == n) {
				//found trivial factor n of n
				return;
			} else {
				//found a real factor of n
				*factor_out = g;

				//stop all other threads
				*factor_not_found_dev = false;
				return;
			}
		}
	}
}
