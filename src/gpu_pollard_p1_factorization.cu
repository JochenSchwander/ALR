#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "gpu_math_stuff.h"
#include "device.h"
#include <stdbool.h>

#define GPU_POLLARD_P1_V1
//#define GPU_POLLARD_P1_V2

#ifdef MACBOOK
int gridSize = 3;
int blockSize = 976;
#endif

#ifdef XMG
int gridSize = 3;
int blockSize = 976;
#endif

//Weichen
//#define DEBUG_GPU_ONLY_CALC

#ifdef DEBUG_GPU_ONLY_CALC
#include <stdio.h>
#include <time.h>
clock_t start, end;
#endif

void setGridSize(int size) {
	gridSize = size;
}

int getGridSize() {
	return gridSize;
}

void setBlockSize(int size) {
	blockSize = size;
}

int getBlockSize() {
	return blockSize;
}

void gpu_pollard_p1_factorization(long long int n, long long int* p, long long int* q, unsigned long int *primes, unsigned long int primes_length) {
	// pointers needed on device
	long long int a, a_max = 1000;
	long long int *n_dev, *p_dev, *a_dev;
	unsigned long int *primes_dev, *primes_length_dev;
	bool factor_not_found = true;
	bool *factor_not_found_dev;


	// allocate memory on device
	hipMalloc((void **) &n_dev, sizeof(long long int));
	hipMalloc((void **) &p_dev, sizeof(long long int));
	hipMalloc((void **) &primes_length_dev, sizeof(unsigned long int));
	hipMalloc((void **) &primes_dev, sizeof(unsigned long int) * primes_length);
	hipMalloc((void **) &a_dev, sizeof(long long int));
	hipMalloc((void **) &factor_not_found_dev, sizeof(bool));

	
	// copy input to device
	hipMemcpy(n_dev, &n, sizeof(long long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_length_dev, &primes_length, sizeof(unsigned long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_dev, primes, sizeof(unsigned long int) * primes_length, hipMemcpyHostToDevice);
	hipMemcpy(factor_not_found_dev, &factor_not_found, sizeof(bool), hipMemcpyHostToDevice);

	for (a = 2; a < a_max && factor_not_found; a++) {
		// copy a in
		hipMemcpy(a_dev, &a, sizeof(long long int), hipMemcpyHostToDevice);

#ifdef DEBUG_GPU_ONLY_CALC
		//measure gpu calculation only
		start = clock();
#endif

		// calculate a prime factor on gpu
		gpu_pollard_p1_factor<<<gridSize,blockSize>>>(n_dev, a_dev, primes_dev, primes_length_dev, p_dev, factor_not_found_dev);
		hipDeviceSynchronize();

#ifdef DEBUG_GPU_ONLY_CALC
		end = clock();
		printf("---> gpu calculation: %lf; a = %lld\n",  (end-start)/(double)CLOCKS_PER_SEC, a);
#endif

		// check if factor allready found
		hipMemcpy(&factor_not_found, factor_not_found_dev, sizeof(bool), hipMemcpyDeviceToHost);
	}

	// copy result to host
	hipMemcpy(p, p_dev, sizeof(long long int), hipMemcpyDeviceToHost);

	// free memory on device
	hipFree(n_dev);
	hipFree(p_dev);
	hipFree(primes_length_dev);
	hipFree(primes_dev);
	hipFree(a_dev);


	// calculate other factor on cpu
	*q = n / *p;
}

__global__ void gpu_pollard_p1_factor(long long int *n_in, long long int *a_in, unsigned long int *primes, unsigned long int *primes_length_in, long long int *factor_out, bool *factor_not_found_dev) {
	unsigned int b;
	const unsigned int b_max = 1000000;
	long long int  e, p, i, g;
	long long int n = *n_in;
	//long long int a = *a_in;
	unsigned long int primes_length = *primes_length_in;

	for (b = 2 + blockIdx.x * blockDim.x + threadIdx.x; b < b_max && *factor_not_found_dev; b += blockDim.x * gridDim.x) {

		//calculate e
#ifdef GPU_POLLARD_P1_V2
		e = 1;
#endif
#ifdef GPU_POLLARD_P1_V1
		e = *a_in;
#endif
		for (i = 0; i < primes_length; i++) {
			p = (long long int) primes[i];
			if (b >= p) {
#ifdef GPU_POLLARD_P1_V2
				e *= gpu_power_mod(p, log((double)b) / log((double) p), n);
#endif
#ifdef GPU_POLLARD_P1_V1
				e = gpu_power_mod(e, p, n);
#endif
			} else {
				break;
			}
		}

		//check if g is a factor of n
#ifdef GPU_POLLARD_P1_V2
		g = gpu_euclidean_gcd(gpu_power_mod(*a_in, e - 1, n), n);
#endif
#ifdef GPU_POLLARD_P1_V1
		g = gpu_euclidean_gcd(e - 1, n);
#endif
		if (g > 1) {
			if (g == n) {
				//found trivial factor n of n
				return;
			} else {
				//found a real factor of n
				*factor_out = g;

				//stop all other threads
				*factor_not_found_dev = false;
				return;
			}
		}
	}
}
