#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "gpu_math_stuff.h"

//#define GPU_POLLARD_P1_V2
#define GPU_POLLARD_P1_V1

void gpu_pollard_p1_factorization(long long int n, long long int* p, long long int* q, unsigned long int *primes, unsigned long int primes_length) {
	// pointers needed on device
	long long int *n_dev, *p_dev;
	unsigned long int *primes_dev, *primes_length_dev;

	// allocate memory on device
	hipMalloc((void **) &n_dev, sizeof(long long int));
	hipMalloc((void **) &p_dev, sizeof(long long int));
	hipMalloc((void **) &primes_length_dev, sizeof(unsigned long int));
	hipMalloc((void **) &primes_dev, sizeof(unsigned long int) * primes_length);
	
	// copy input to device
	hipMemcpy(n_dev, &n, sizeof(long long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_length_dev, &primes_length, sizeof(unsigned long int), hipMemcpyHostToDevice);
	hipMemcpy(primes_dev, primes, sizeof(unsigned long int) * primes_length, hipMemcpyHostToDevice);

	// calculate a prime factor on gpu
	gpu_pollard_p1_factor<<<1,1>>>();
	hipDeviceSynchronize();

	// copy result to host
	hipMemcpy(p, p_dev, sizeof(long long int), hipMemcpyDeviceToHost);

	// free memory on device
	hipFree(n_dev);
	hipFree(p_dev);
	hipFree(primes_length_dev);
	hipFree(primes_dev);

	// calculate other factor on cpu
	*q = n / *p;
}

__global__ void gpu_pollard_p1_factor(long long int n, unsigned long int *primes, unsigned long int primes_length, long long int *factor) {
	long long int b_max = 1000000;
	long long int a_max = 1000;
	long long int b, e, p, i, a, g;

	for (a = 2; a < a_max; a++) {

		for (b = 2; b < b_max; b++) {

			//calculate e
#ifdef GPU_POLLARD_P1_V2
			e = 1;
#endif
#ifdef GPU_POLLARD_P1_V1
			e = a;
#endif
			for (i = 0; i < primes_length; i++) {
				p = (long long int) primes[i];
				if (b >= p) {
#ifdef GPU_POLLARD_P1_V2
					e *= power_mod(p, log((long double)b) / log((long double) p), n);
#endif
#ifdef GPU_POLLARD_P1_V1
					e = power_mod(e, p, n);
#endif
				} else {
					break;
				}
			}

			//check if g is a factor of n
#ifdef GPU_POLLARD_P1_V2
			g = euclidean_gcd(power_mod(a, e - 1, n), n);
#endif
#ifdef GPU_POLLARD_P1_V1
			g = euclidean_gcd(e - 1, n);
#endif
			if (g > 1) {
				if (g == n) {
					//found trivial factor n of n
					break;
				} else {
					//found a real factor of n
					*factor = g;
					return;
				}
			}
		}
	}
}
