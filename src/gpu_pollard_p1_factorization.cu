#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "gpu_math_stuff.h"




void pollard_p1_factorization(long long int n, long long int* p, long long int* q) {
	
	
	
	//TODO
	*p = pollard_p1_factor(n);
	*q = n / *p;
}


__global__ long long int pollard_p1_factor(long long int n, long long int *factor, long long int a) {
	long long int bound = n / 2; //quadratwurzel oder anders kleiner?
	long long int a, i, mult, b;

	for (a = 2; a < bound; a++) {

		mult = a;
		for (i = 1; i < bound; i++) {

			mult = power_mod(mult, i, n);

			b = euclidean_gcd(mult - 1, n);

			if (b > 1) {
				if (b == n) {
					break;
				} else {
					return b;
				}
			}

		}
	}

	return n;
}
