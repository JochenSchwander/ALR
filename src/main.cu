#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

//#define DEBUG_PRINT_CLOCKS


void read_primes(unsigned long int *primes);

int main() {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);

	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));


	
	read_primes(primes);

	*n = 902491;
	e = 5;


	printf("n = %lld\n", *n);
	start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
#endif
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %lld\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, sizeof(long long int),hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *host_p, *host_q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *host_p, *host_q, (end-start)	/(double)CLOCKS_PER_SEC);
#endif

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);
	system("say martin ist ein bob!");
	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0; 
	
	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%u,", &prime)) != EOF) { 
		primes[count++] = prime; 
	} 
	fclose(datei); 
}
