#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdint.h>
#include <stdlib.h>
#include <stdio.h>

void read_primes(uint16_t *primes, uint16_t primes_length);

int main() {
	uint16_t primes_length = 78499;
	uint16_t *primes = (uint16_t *) malloc(sizeof(uint16_t) * primes_length);
	read_primes(primes, primes_length);

	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	int size = sizeof(long long int);
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));


	*n = 902491;
	e = 5;


	printf("n = %lld\n", *n);
	start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %lld\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, size,hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

	printf("p = %lld; q = %lld in %lf seconds\n", *host_p, *host_q, (end-start)/(double)CLOCKS_PER_SEC);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	return 0;
}

void read_primes(uint16_t *primes, uint16_t primes_length) {
	FILE *datei;
	int prime;
	int count = 0; 
	/* Zum Lesen �ffnen */ 
	datei = fopen("src/primzahlenbis1millionen.txt", "r");
	while ((fscanf(datei, "%d,", &prime)) != EOF) { 
		primes[count++] = (uint16_t) prime; 
	} 
	fclose(datei); // Loop over strings
	for (int i = 0; i < 78499; i++) { 
		printf("%d \n", primes[i]);
	}
}
