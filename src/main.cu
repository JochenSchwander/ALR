#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include "statistic_definitions.h"

void read_primes(unsigned long int *primes);

int main(int argc, char *argv[]) {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);
	int i, j;
	bool statisticMode = false;

	//time measurement
	clock_t start, end;

	long long int *p, *q, *n, e, d;
	n = (long long int*) malloc(sizeof(long long int));
	p = (long long int*) malloc(sizeof(long long int));
	q = (long long int*) malloc(sizeof(long long int));

	read_primes(primes);

	int choice;
	double cpuTime, gpuTime;
	bool isEnd = false;

	if (argc > 1) {
		if (strstr(argv[1], "-statistic") != NULL) {
			statisticMode = true;
			choice = 7;
			goto menu;
		}
	}

	while (!isEnd) {
		printf("------------- Menu ----------------\n");
		printf("1. CPU & GPU - Starten mit Standard n und e ...\n");
		printf("2. CPU & GPU - Eingabe von n und e ...\n");
		printf("3. CPU - Starten mit Standard n und e ...\n");
		printf("4. CPU - Eingabe von n und e ...\n");
		printf("5. GPU - Starten mit Standard n ...\n");
		printf("6. GPU - Eingabe von n ...\n");
		printf("7. GPU - BlockSize/GridSize Statistik ...\n");
		printf("8. CPU & GPU - n's aus Datei einlesen und Statistik erstellen ...\n");
		printf("0. Programm verlassen ...\n");
		printf("Eingabe Menuepunkt: ");
		scanf("%d", &choice);

menu: 
		//*n = 65521LL * 65537LL;  //n6, biggest possible n
		//*n = 57037LL * 57041LL;  //n5
		//*n = 40709LL * 40739LL;  //n4
		//*n = 32621LL * 32633LL;  //n3!
		//*n = 25087LL * 25097LL;  //n2
		*n = 20903LL * 20921LL;  //n1
		//*n = 7331LL * 7333LL;
		//*n = 902491;
		e = 21;

		*p = 1;
		*q = 1;

		switch (choice) {
		case 1:
			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU erchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", cpuTime, (unsigned long) (end - start), *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);

			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", gpuTime, (unsigned long) (end - start), *p, *q);

			printf("---------------------------\n");
			if (cpuTime > gpuTime) {
				printf("GPU war %lf Sekunden schneller\n", cpuTime - gpuTime);
				printf("GPU war %lf mal schneller\n", cpuTime / gpuTime);
			} else {
				printf("CPU war %lf Sekunden schneller\n", gpuTime - cpuTime);
				printf("CPU war %lf mal schneller\n", gpuTime / cpuTime);
			}
			break;
		case 2:
			printf("Eingabe n: ");
			scanf("%lld", n);
			printf("Eingabe e: ");
			scanf("%lld", &e);

			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", cpuTime, (unsigned long) (end - start), *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);

			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", gpuTime, (unsigned long) (end - start), *p, *q);

			printf("---------------------------\n");
			if (cpuTime > gpuTime) {
				printf("GPU war %lf Sekunden schneller\n", cpuTime - gpuTime);
				printf("GPU war %lf mal schneller\n", cpuTime / gpuTime);
			} else {
				printf("CPU war %lf Sekunden schneller\n", gpuTime - cpuTime);
				printf("CPU war %lf mal schneller\n", gpuTime / cpuTime);
			}
			break;
		case 3:
			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", cpuTime, (unsigned long) (end - start), *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);
			break;
		case 4:
			printf("Eingabe n: ");
			scanf("%lld", n);
			printf("Eingabe e: ");
			scanf("%lld", &e);

			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", cpuTime, (unsigned long) (end - start), *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);
			break;
		case 5:
			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", gpuTime, (unsigned long) (end - start), *p, *q);
			break;
		case 6:
			printf("Eingabe n: ");
			scanf("%lld", n);
			
			printf("------------- Eingabe -------------\n");
			printf("n = %lld, e = %lld\n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("Ergebnis nach %lf Sekunden / %lu clocks: \np = %lld\nq = %lld \n", gpuTime, (unsigned long) (end - start), *p, *q);
			break;
		case 7: //first run takes longer, remove from statistics
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);

			printf("gridSize;blockSize;p;q;clocks;seconds\n");
			for (i = STATISTIC_MULTIPROCESSORS; i <= STATISTIC_MAX_GRIDSIZE; i += STATISTIC_MULTIPROCESSORS) {
				setGridSize(i);
				for (j = STATISTIC_BLOCKSIZE_STEPSIZE; j <= STATISTIC_MAX_BLOCKSIZE; j += STATISTIC_BLOCKSIZE_STEPSIZE) {
					if ((i / STATISTIC_MULTIPROCESSORS) * j > STATISTIC_MAX_THREADS_PER_MULTIPROCESSOR) {
						continue;
					}
					setBlockSize(j);
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
					printf("%d;%d;%lld;%lld;%lu;%lf\n", getGridSize(), getBlockSize(), *p, *q, (unsigned long) (end - start), gpuTime);
					*p = 1;
					*q = 1;
				}
			}
			if (statisticMode) {
				isEnd = true;
			}
			break;
		case 8: {
			FILE *input, *statOutput; //*output
			char filename[50];
			char buff[25];
			input = fopen("fileofN_check.txt", "r");
			//NOT IN SUBFOLDER "statistic" -> program crashes if folder isn't there...
			//output = fopen("outputCalculation.txt", "a+");

			//open and create statistic output file for excel import
			time_t timeforFilename = time(0);
			strftime(buff, 25, "%Y%m%d_%H_%M_%S", localtime(&timeforFilename));
			//NOT IN SUBFOLDER "statistic" -> program crashes if folder isn't there...
			sprintf(filename, "statOutput_%s.csv", buff);
			statOutput = fopen(filename, "w");

			//fprintf(output, "_____________________________________________________________________________________________________________________________________________________________________________\n");
			//fprintf(output, " 	n		|	 	TimeStamp		 |		  CPU(p,q)		   |		CPU time		|		  GPU(p,q)		   |		  GPU time		|						Result		\n");

			// read n's out of file and calculate
			while ((fscanf(input, "%lld,", n)) != EOF) {
				// log n to output
				printf("%lld		", *n);
				//fprintf(output, "%lld		", *n);
				// log n to statOutput
				fprintf(statOutput, "%lld;", *n);
				//timestamp output
				//time_t nowtime = time(0);
				//strftime(buff, 25, "%Y-%m-%d %H:%M:%S", localtime(&nowtime));
				//printf("%s		", buff);
				//fprintf(output, "%s		", buff);
				// CPU calculation
				start = clock();
				pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				// log result of p and q
				printf("(C) p=%lld, q=%lld		", *p, *q);
				//fprintf(output, "(C) p=%lld, q=%lld		", *p, *q);
				cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
				// log result of CPU and time
				printf("%lf Sekunden		", cpuTime);
				//fprintf(output, "%lf Sekunden		", cpuTime);
				// log CPU time to statOutput
				fprintf(statOutput, "%lf;", cpuTime);
				// GPU calculation
				start = clock();
				gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				// log result of p and q
				printf("(G) p=%lld, q=%lld		", *p, *q);
				//fprintf(output, "(G) p=%lld, q=%lld		", *p, *q);
				gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
				// log result of GPU and time
				printf("%lf Sekunden		", gpuTime);
				//fprintf(output, "%lf Sekunden		", gpuTime);
				// log CPU time to statOutput
				fprintf(statOutput, "%lf;", gpuTime);
				// log result of CPU and GPU, calculate which is faster
				if (cpuTime > gpuTime) {
					printf("GPU %lf Sekunden | %lf mal schneller\n", cpuTime - gpuTime, cpuTime / gpuTime);
					//fprintf(output, "GPU %lf Sekunden | %lf mal schneller\n", cpuTime - gpuTime, cpuTime / gpuTime);
				} else {
					printf("CPU %lf Sekunden | %lf mal schneller\n", gpuTime - cpuTime, gpuTime / cpuTime);
					//fprintf(output, "CPU %lf Sekunden | %lf mal schneller\n", gpuTime - cpuTime, gpuTime / cpuTime);
				}
				fprintf(statOutput, "%lld;%lld;\n", *p, *q);
				printf("\n");
				//fprintf(output, "\n");

				//reset p and q just to be save
				*p = 1;
				*q = 1;
			}

			fclose(input);
			//fclose(output);
			fclose(statOutput);
		}
			break;
		default:
			isEnd = true;
			break;
		}
	}

	free(n);
	free(p);
	free(q);

	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0;

	datei = fopen("primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime;
	}
	fclose(datei);
}
