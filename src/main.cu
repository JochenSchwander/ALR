#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "factorization.h"
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

//#define DEBUG_PRINT_CLOCKS


void read_primes(unsigned long int *primes);

int main() {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);

	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));
	
	read_primes(primes);

	*n = 902491;
	e = 5;

	int choice;

	printf("------------- Menu ----------------\n");
	printf("1. starten mit Standard n und e ...\n");
	printf("2. Eingabe von n und e ...\n");
	printf("Eingabe choice: ");
	scanf("%d",&choice);

	switch(choice){
		case 1:	printf("------------- Ausgabe -------------\n");
				printf("========= CPU ========\n");
				printf("CPU berchnung wird gestartet...\n");
				start = clock();
				pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				double cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
				printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
				printf("Ergebnis nach (%lf) Sekunden : p = %lld; q = %lld \n", cpuTime, *p, *q);
				d = calculatePrivateKey(e,*p,*q);
				printf("d = %lld\n", d);

				printf("========= GPU ========\n");
				printf("GPU Register werden beschrieben\n");
				printf("GPU berechnung wird gestartet");
				start = clock();
				gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				double gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
				printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
				printf("Ergebnis nach (%lf) Sekunden : p = %lld; q = %lld \n", gpuTime, *p, *q);

				printf("---------------------------\n")
				printf("GPU war (%lf) Sekunden schneller", cpuTime-gpuTime);
				if(cpuTime > gpuTime) {
					printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
				} else {
					printf("CPU war % mal schneller\n", gpuTime/cpuTime);
				}
			break;
		case 2:	printf("Eingabe n: ");
				scanf("%lld",n);
				printf("Eingabe e: ");
				scanf("%lld",&e);
				printf("You input n=%lld und e=%lld \n", *n, e);

				printf("------------- Ausgabe -------------\n");
				printf("========= CPU ========\n");
				printf("CPU berchnung wird gestartet...\n");
				start = clock();
				double cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
				pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
				printf("Ergebnis nach (%lf) Sekunden : p = %lld; q = %lld \n", cpuTime, *p, *q);
				d = calculatePrivateKey(e,*p,*q);
				printf("d = %lld\n", d);

				printf("========= GPU ========\n");
				printf("GPU Register werden beschrieben\n");
				printf("GPU berechnung wird gestartet");
				start = clock();
				double gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
				gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
				printf("Ergebnis nach (%lf) Sekunden : p = %lld; q = %lld \n", gpuTime, *p, *q);

				printf("---------------------------\n")
				printf("GPU war (%lf) Sekunden schneller", cpuTime-gpuTime);
				if(cpuTime > gpuTime) {
					printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
				} else {
					printf("CPU war % mal schneller\n", gpuTime/cpuTime);
				}
			break;
		default:	// do nothing
			break;
	}

	/*start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
#endif
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %lld\n", d);*/


	/*//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, sizeof(long long int),hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *host_p, *host_q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *host_p, *host_q, (end-start)	/(double)CLOCKS_PER_SEC);
#endif

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	*/

	/*
	start = clock();
	//factorization(*n, p, q);
	gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
#endif*/

	system("say das programm wurde erfolgreich ausgefuehrt und martin ist kein ein bob!");
	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0; 
	
	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime; 
	} 
	fclose(datei); 
}
