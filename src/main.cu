#include "stdio.h"
#include "factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>

//CUDA section
#define N 10
//END CUDA section

int main()
{
	//CUDA
	long *dev_n, *dev_p, *dev_q;
	int size = sizeof(long int);
	long p, q, n, e, d, expectedD = 185;
	n = 989;
	e = 5;

	printf("n = %ld\n", n);
	factorization(n, &p, &q);
	printf("p = %ld; q = %ld\n", p, q);
	d = calculatePrivateKey(e,p,q);
	printf("d = %ld\n", d);
	printf("lala");
	if (expectedD == d)
		printf("geknackt!");




	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, N * sizeof(long));
	hipMalloc((void **) &dev_p, N * sizeof(long));
	hipMalloc((void **) &dev_q, N * sizeof(long));

	hipMemcpy( dev_n, &n, size,hipMemcpyHostToDevice);

	printf("utz utz2");
	gpu_factorization<<<N,1>>>(*dev_n, dev_p, dev_q);

	printf("utz utz1");
	hipDeviceSynchronize();
	printf("utz utz");
	hipMemcpy( &p, dev_p, sizeof(long),	hipMemcpyDeviceToHost);
	hipMemcpy( &q, dev_q, sizeof(long),	hipMemcpyDeviceToHost);

	printf("p = %ld; q = %ld\n", p, q);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	return 0;
}
