#include "hip/hip_runtime.h"
#include "stdio.h"
#include "factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>



int main()
{
	//CUDA
	long *dev_n, *dev_p, *dev_q;
	int size = sizeof(long);
	//time measurement
	clock_t start, end;

	long *p, *host_p, *host_q, *q, *n, e, d;
	p = (long*)malloc(sizeof(long));
	q = (long*)malloc(sizeof(long));
	n = (long*)malloc(sizeof(long));

	host_p = (long*)malloc(sizeof(long));
	host_q = (long*)malloc(sizeof(long));


	*n = 902491;
	e = 5;


	printf("n = %ld\n", *n);
	start = clock();
	factorization(*n, p, q);
	end = clock();
	printf("p = %ld; q = %ld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %ld\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long));
	hipMalloc((void **) &dev_p, sizeof(long));
	hipMalloc((void **) &dev_q, sizeof(long));

	hipMemcpy( dev_n, n, size,hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long),	hipMemcpyDeviceToHost);

	printf("p = %ld; q = %ld in %lf seconds\n", *host_p, *host_q, (end-start)/(double)CLOCKS_PER_SEC);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	system("say martin ist ein bob.");
	return 0;
}
