#include "hip/hip_runtime.h"
#include "stdio.h"
//#include "factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <time.h>



int main()
{
	//CUDA
	__int64 *dev_n, *dev_p, *dev_q;
	int size = sizeof(__int64);
	//time measurement
	clock_t start, end;

	__int64 *p, *host_p, *host_q, *q, *n, e, d;
	p = (__int64*)malloc(sizeof(__int64));
	q = (__int64*)malloc(sizeof(__int64));
	n = (__int64*)malloc(sizeof(__int64));

	host_p = (__int64*)malloc(sizeof(__int64));
	host_q = (__int64*)malloc(sizeof(__int64));


	*n = 902491;
	e = 5;


	printf("n = %I64d\n", *n);
	start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q);
	end = clock();
	printf("p = %I64d; q = %I64d in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %I64d\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(__int64));
	hipMalloc((void **) &dev_p, sizeof(__int64));
	hipMalloc((void **) &dev_q, sizeof(__int64));

	hipMemcpy( dev_n, n, size,hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(__int64),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(__int64),	hipMemcpyDeviceToHost);

	printf("p = %I64d; q = %I64d in %lf seconds\n", *host_p, *host_q, (end-start)/(double)CLOCKS_PER_SEC);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	return 0;
}
