#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "factorization.h"
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

//#define DEBUG_PRINT_CLOCKS


void read_primes(unsigned long int *primes);

int main() {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);

	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));
	
	read_primes(primes);

	*n = 902491;
	e = 5;

	int choice;
	double cpuTime, gpuTime;
	bool isEnd = false;

	// TODO add menu point for GPU and CPU calculation seperat
	while(!isEnd){
		printf("------------- Menu ----------------\n");
		printf("1. CPU & GPU - starten mit Standard n und e ...\n");
		printf("2. CPU & GPU - Eingabe von n und e ...\n");
		printf("3. CPU - starten mit Standard n und e ...\n");
		printf("4. CPU - Eingabe von n und e ...\n");
		printf("5. GPU - starten mit Standard n ...\n");
		printf("6. GPU - Eingabe von n ...\n");
		printf("7. Exit the program ...\n");
		printf("Eingabe choice: ");
		scanf("%d",&choice);

		switch(choice){
			case 1:	printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 2:	printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld/nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : /np = %lld/nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 3: printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 4: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 5: printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);
				break;
			case 6: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("You input n=%lld\n", *n);

					printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld/nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : /np = %lld/nq = %lld \n", gpuTime, *p, *q);
				break;
			default: isEnd = true;
				break;
		}
	}

	/*start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
#endif
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %lld\n", d);*/


	/*//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, sizeof(long long int),hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *host_p, *host_q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *host_p, *host_q, (end-start)	/(double)CLOCKS_PER_SEC);
#endif

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	*/

	/*
	start = clock();
	//factorization(*n, p, q);
	gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
#ifdef DEBUG_PRINT_CLOCKS
	printf("p = %lld; q = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
#else
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
#endif*/

	system("say das programm wurde erfolgreich ausgefuehrt und martin ist kein kein kein bob! Notiz an Phil /!");
	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0; 
	
	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime; 
	} 
	fclose(datei); 
}
