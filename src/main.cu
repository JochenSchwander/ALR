#include "hip/hip_runtime.h"
#define __HIPCC__

#include <stdio.h>
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include "mpz/mpz.h"
#include "kernel.h"



void read_primes(unsigned long int *primes);


int main() {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);

	//time measurement
	clock_t start, end;

	long long int *p, *q, *n, e, d;
	n = (long long int*)malloc(sizeof(long long int));
	
	read_primes(primes);

	*n = 20903LL * 20921LL;
	//*n = 7331LL * 7333LL;
	//*n = 902491;
	e = 5;

	int choice;
	double cpuTime, gpuTime;
	bool isEnd = false;


	// TODO add menu point for GPU and CPU calculation seperat
	while(!isEnd){



		p = (long long int*)malloc(sizeof(long long int));
		q = (long long int*)malloc(sizeof(long long int));

		//system("say Bitte waehlen sie einen Menuepunkt. Vergiss nicht, martin ist ein bob/!");
		printf("------------- Menu ----------------\n");
		printf("1. CPU & GPU - starten mit Standard n und e ...\n");
		printf("2. CPU & GPU - Eingabe von n und e ...\n");
		printf("3. CPU - starten mit Standard n und e ...\n");
		printf("4. CPU - Eingabe von n und e ...\n");
		printf("5. GPU - starten mit Standard n ...\n");
		printf("6. GPU - Eingabe von n ...\n");
		printf("7. Exit the program ...\n");
		printf("Eingabe choice: ");
		scanf("%d",&choice);

		switch(choice){
			case 1:	printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 2:	printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 3: printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 4: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 5: printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);
				break;
			case 6: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("You input n=%lld\n", *n);

					printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);
				break;
			default: isEnd = true;
				break;
		}
		free(p);
		free(q);
	}

	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0; 
	
	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime; 
	} 
	fclose(datei); 
}
