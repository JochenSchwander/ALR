#include "hip/hip_runtime.h"
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

void read_primes(unsigned long int *primes);

int main(int argc, char *argv[]) {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(sizeof(unsigned long int) * primes_length);
	int i, j;
	bool statisticMode = false;

	//time measurement
	clock_t start, end;

	long long int *p, *q, *n, e, d;
	n = (long long int*)malloc(sizeof(long long int));
	
	read_primes(primes);

	*n = 20903LL * 20921LL;
	//*n = 7331LL * 7333LL;
	//*n = 902491;
	e = 5;

	int choice;
	double cpuTime, gpuTime;
	bool isEnd = false;

	if (argc > 1) {
		if (strstr(argv[1], "-statistic") != NULL) {
			statisticMode = true;
			choice = 7;
			goto menu;
		}
	}

	// TODO add menu point for GPU and CPU calculation seperat
	while(!isEnd){
		//system("say Bitte waehlen sie einen Menuepunkt. Vergiss nicht, martin ist ein bob/!");
		printf("------------- Menu ----------------\n");
		printf("1. CPU & GPU - starten mit Standard n und e ...\n");
		printf("2. CPU & GPU - Eingabe von n und e ...\n");
		printf("3. CPU - starten mit Standard n und e ...\n");
		printf("4. CPU - Eingabe von n und e ...\n");
		printf("5. GPU - starten mit Standard n ...\n");
		printf("6. GPU - Eingabe von n ...\n");
		printf("7. GPU - BlockSize/GridSize Statistik\n");
		printf("8. Exit the program ...\n");
		printf("Eingabe choice: ");
		scanf("%d",&choice);

menu:
		p = (long long int*)malloc(sizeof(long long int));
		q = (long long int*)malloc(sizeof(long long int));

		switch(choice){
			case 1:	printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 2:	printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);

					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);

					printf("---------------------------\n");
					if(cpuTime > gpuTime) {
						printf("GPU war %lf Sekunden schneller\n", cpuTime-gpuTime);
						printf("GPU war %lf mal schneller\n", cpuTime/gpuTime);
					} else {
						printf("CPU war %lf Sekunden schneller\n", gpuTime-cpuTime);
						printf("CPU war %lf mal schneller\n", gpuTime/cpuTime);
					}
				break;
			case 3: printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 4: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("Eingabe e: ");
					scanf("%lld",&e);
					printf("You input n=%lld und e=%lld \n", *n, e);

					printf("------------- Ausgabe -------------\n");
					printf("========= CPU ========\n");
					printf("CPU berchnung wird gestartet...\n");
					start = clock();
					pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					cpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n", cpuTime, *p, *q);
					d = calculatePrivateKey(e,*p,*q);
					printf("d = %lld\n", d);
				break;
			case 5: printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);
				break;
			case 6: printf("Eingabe n: ");
					scanf("%lld",n);
					printf("You input n=%lld\n", *n);

					printf("------------- Ausgabe -------------\n");
					printf("========= GPU ========\n");
					printf("GPU Register werden beschrieben\n");
					printf("GPU berechnung wird gestartet\n");
					start = clock();
					gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
					end = clock();
					gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
					printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q, (unsigned long)(end-start));
					printf("Ergebnis nach (%lf) Sekunden : \np = %lld\nq = %lld \n", gpuTime, *p, *q);
				break;
			case 7: printf("gridSize,blockSize,p,q,clocks,seconds\n");
					for (i = 1; i <= 1024; i *= 2) {
						setGridSize(i);
						for (j = 32; j <= 1024; j += 32) {
							if (i * j > 16*640) {
								continue;
							}
							setBlockSize(j);
							printf("%d,%d,", getGridSize(), getBlockSize());
							start = clock();
							gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
							end = clock();
							gpuTime = (end-start)/(double)CLOCKS_PER_SEC;
							printf("%lld,%lld,%lu,%lf\n", *p, *q, (unsigned long)(end-start), gpuTime);
							*p = 1;
							*q = 1;
						}
					}
					if (statisticMode) {
						isEnd = true;
					}
				break;
			default: isEnd = true;
				break;
		}
		free(p);
		free(q);
	}

	return 0;
}

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0; 
	
	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime; 
	} 
	fclose(datei); 
}
