#include "hip/hip_runtime.h"
#include <stdio.h>
//#include "factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

void read_primes(unsigned int *primes);

int main() {
	unsigned int primes_length = 78498;
	unsigned int *primes = (unsigned int *) malloc(sizeof(unsigned int) * primes_length);

	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));


	
	read_primes(primes);

	*n = 902491;
	e = 5;


	printf("n = %lld\n", *n);
	start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q, primes, primes_length);
	end = clock();
	printf("p = %lld; q = %lld in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %lld\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, sizeof(long long int),hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

	printf("p = %lld; q = %lld in %lf seconds\n", *host_p, *host_q, (end-start)/(double)CLOCKS_PER_SEC);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	return 0;
}

void read_primes(unsigned int *primes) {
	FILE *datei;
	unsigned int prime;
	int count = 0; 
	/* Zum Lesen �ffnen */ 
	datei = fopen("src/primzahlenbis1millionen.txt", "r");
	while ((fscanf(datei, "%u,", &prime)) != EOF) { 
		primes[count++] = prime; 
	} 
	fclose(datei); 
	// Loop over strings
	for (int i = 0; i < 78498; i++) { 
		printf("%u \n", primes[i]);
	}
}
