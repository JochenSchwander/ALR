#include "hip/hip_runtime.h"
#include "stdio.h"
//#include "factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <time.h>



int main()
{
	//CUDA
	long long int *dev_n, *dev_p, *dev_q;
	int size = sizeof(long long int);
	//time measurement
	clock_t start, end;

	long long int *p, *host_p, *host_q, *q, *n, e, d;
	p = (long long int*)malloc(sizeof(long long int));
	q = (long long int*)malloc(sizeof(long long int));
	n = (long long int*)malloc(sizeof(long long int));

	host_p = (long long int*)malloc(sizeof(long long int));
	host_q = (long long int*)malloc(sizeof(long long int));


	*n = 902491;
	e = 5;


	printf("n = %I64d\n", *n);
	start = clock();
	//factorization(*n, p, q);
	pollard_p1_factorization(*n, p, q);
	end = clock();
	printf("p = %I64d; q = %I64d in %lf seconds\n", *p, *q, (end-start)/(double)CLOCKS_PER_SEC);
	d = calculatePrivateKey(e,*p,*q);
	printf("d = %I64d\n", d);


	//allocate the momory on th GPU
	hipMalloc((void **) &dev_n, sizeof(long long int));
	hipMalloc((void **) &dev_p, sizeof(long long int));
	hipMalloc((void **) &dev_q, sizeof(long long int));

	hipMemcpy( dev_n, n, size,hipMemcpyHostToDevice);

	start = clock();
	gpu_factorization<<<4,384>>>(dev_n, dev_p, dev_q);
	hipDeviceSynchronize();
	end = clock();

	hipMemcpy( host_p, dev_p, sizeof(long long int),	hipMemcpyDeviceToHost);
	hipMemcpy( host_q, dev_q, sizeof(long long int),	hipMemcpyDeviceToHost);

	printf("p = %I64d; q = %I64d in %lf seconds\n", *host_p, *host_q, (end-start)/(double)CLOCKS_PER_SEC);

	hipFree(dev_p);
	hipFree(dev_q);
	hipFree(dev_n);

	return 0;
}
