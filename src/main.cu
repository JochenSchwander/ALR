#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu_pollard_p1_factorization.h"
#include "pollard_p1_factorization.h"
#include "gpu_factorization.h"
#include "rsacalculation.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

void read_primes(unsigned long int *primes) {
	FILE *datei;
	unsigned long int prime;
	int count = 0;

	datei = fopen("src/primes.txt", "r");
	while ((fscanf(datei, "%lu,", &prime)) != EOF) {
		primes[count++] = prime;
	}
	fclose(datei);
}

int main() {
	unsigned long int primes_length = 78498;
	unsigned long int *primes = (unsigned long int *) malloc(
			sizeof(unsigned long int) * primes_length);

	//time measurement
	clock_t start, end;

	long long int *p, *q, *n, e, d;
	n = (long long int*) malloc(sizeof(long long int));

	read_primes(primes);

	*n = 902491;
	e = 5;

	int choice;
	double cpuTime, gpuTime;
	bool isEnd = false;

	while (!isEnd) {
		p = (long long int*) malloc(sizeof(long long int));
		q = (long long int*) malloc(sizeof(long long int));

		printf("------------- Menu ----------------\n");
		printf("1. CPU & GPU - starten mit Standard n und e ...\n");
		printf("2. CPU & GPU - Eingabe von n und e ...\n");
		printf("3. CPU - starten mit Standard n und e ...\n");
		printf("4. CPU - Eingabe von n und e ...\n");
		printf("5. GPU - starten mit Standard n ...\n");
		printf("6. GPU - Eingabe von n ...\n");
		printf("7. File input of n's (statistic mode) ...\n");
		printf("99. Exit the program ...\n");
		printf("Eingabe choice: ");
		scanf("%d", &choice);

		switch (choice) {
		case 1:
			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					cpuTime, *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);

			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					gpuTime, *p, *q);

			printf("---------------------------\n");
			if (cpuTime > gpuTime) {
				printf("GPU war %lf Sekunden schneller\n", cpuTime - gpuTime);
				printf("GPU war %lf mal schneller\n", cpuTime / gpuTime);
			} else {
				printf("CPU war %lf Sekunden schneller\n", gpuTime - cpuTime);
				printf("CPU war %lf mal schneller\n", gpuTime / cpuTime);
			}
			break;
		case 2:
			printf("Eingabe n: ");
			scanf("%lld", n);
			printf("Eingabe e: ");
			scanf("%lld", &e);
			printf("You input n=%lld und e=%lld \n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					cpuTime, *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);

			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					gpuTime, *p, *q);

			printf("---------------------------\n");
			if (cpuTime > gpuTime) {
				printf("GPU war %lf Sekunden schneller\n", cpuTime - gpuTime);
				printf("GPU war %lf mal schneller\n", cpuTime / gpuTime);
			} else {
				printf("CPU war %lf Sekunden schneller\n", gpuTime - cpuTime);
				printf("CPU war %lf mal schneller\n", gpuTime / cpuTime);
			}
			break;
		case 3:
			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					cpuTime, *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);
			break;
		case 4:
			printf("Eingabe n: ");
			scanf("%lld", n);
			printf("Eingabe e: ");
			scanf("%lld", &e);
			printf("You input n=%lld und e=%lld \n", *n, e);

			printf("------------- Ausgabe -------------\n");
			printf("========= CPU ========\n");
			printf("CPU berchnung wird gestartet...\n");
			start = clock();
			pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					cpuTime, *p, *q);
			d = calculatePrivateKey(e, *p, *q);
			printf("d = %lld\n", d);
			break;
		case 5:
			printf("------------- Ausgabe -------------\n");
			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					gpuTime, *p, *q);
			break;
		case 6:
			printf("Eingabe n: ");
			scanf("%lld", n);
			printf("You input n=%lld\n", *n);

			printf("------------- Ausgabe -------------\n");
			printf("========= GPU ========\n");
			printf("GPU Register werden beschrieben\n");
			printf("GPU berechnung wird gestartet\n");
			start = clock();
			gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
			end = clock();
			gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
			printf("p = %lld\nq = %lld in %lu clocks\n", *p, *q,
					(unsigned long) (end - start));
			printf("Ergebnis nach %lf Sekunden : \np = %lld\nq = %lld \n",
					gpuTime, *p, *q);
			break;
		case 7:
			FILE *input, *output;
			input = fopen("src/listofNumbers.txt", "r");
			output = fopen("src/outputCalculation.txt", "a+");

			fprintf(output,
					"\nTimeStamp 	|	CPU(p,q)	|	CPU time	|	GPU(p,q)	|	GPU time	|	Result\n");

			// read n's out of file and calculate
			while ((fscanf(input, "%lu,", &n)) != EOF) {
				// timestamp output
				char buff[25];
				time_t now = time(0);
				strftime(buff, 25, "%Y-%m-%d %H:%M:%S", localtime(&now));
				fprintf(output, "%s		", buff);
				// CPU calculation
				start = clock();
				pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				// log result of p and q
				fprintf(output, "C p=%lld, q=%lld		", *p, *q);
				cpuTime = (end - start) / (double) CLOCKS_PER_SEC;
				// log result of CPU and time
				fprintf(output, "%lf Sekunden		", cpuTime);
				// GPU calculation
				start = clock();
				gpu_pollard_p1_factorization(*n, p, q, primes, primes_length);
				end = clock();
				// log result of p and q
				fprintf(output, "G p=%lld, q=%lld		", *p, *q);
				gpuTime = (end - start) / (double) CLOCKS_PER_SEC;
				// log result of GPU and time
				fprintf(output, "%lf Sekunden		", gpuTime);
				// log result of CPU and GPU, calculate which is faster
				if (cpuTime > gpuTime) {
					fprintf(output, "GPU %lf Sekunden | %lf mal schneller\n",
							cpuTime - gpuTime, cpuTime / gpuTime);
				} else {
					fprintf(output, "CPU %lf Sekunden | %lf mal schneller\n",
							gpuTime - cpuTime, gpuTime / cpuTime);
				}
			}
			fclose(input);
			break;
		default:
			isEnd = true;
			break;
		}
		free(p);
		free(q);
	}

	return 0;
}
