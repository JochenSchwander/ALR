#include "hip/hip_runtime.h"
#include "gpu_factorization.h"

__global__ void gpu_factorization(long* n, long* p, long* q) {


	long local_n = *n;
	long i;
	long idx = blockIdx.x * blockDim.x + threadIdx.x;
	long step_size = blockDim.x*gridDim.x;
	long steps = local_n/2;
	//TODO: n = sqrt(n) !

	for (i=3+idx; i<steps; i += step_size)
	{
		if(local_n % i == 0)
		{
			*p = i;
		}
	}
	*q = local_n / *p;

	__syncthreads();
}
