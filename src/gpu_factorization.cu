#include "hip/hip_runtime.h"
#include "gpu_factorization.h"

__global__ void gpu_factorization(long n, long* p, long* q) {



	long i;
	long idx = blockIdx.x * blockDim.x + threadIdx.x;
	long step_size = blockDim.x*gridDim.x;

	//TODO: n = sqrt(n) !
	for (i=idx; i<n; i += step_size)
	{
		if(n % i == 0)
		{
			*p = i;
		}
	}
	*q = n / *p;

	__syncthreads();
}
