#include "hip/hip_runtime.h"
#include "gpu_factorization.h"

__global__ void gpu_factorization(long long int* n, long long int* p, long long int* q) {


	long long int local_n = *n;
	long long int i;
	long long int idx = blockIdx.x * blockDim.x + threadIdx.x;
	long long int step_size = blockDim.x*gridDim.x;
	long long int steps = local_n/2;
	//TODO: n = sqrt(n) !

	for (i=3+idx; i<steps; i += step_size)
	{
		if(local_n % i == 0)
		{
			*p = i;
		}
	}
	*q = local_n / *p;

	__syncthreads();
}
