#include "hip/hip_runtime.h"
#include "gpu_factorization.h"

__global__ void gpu_factorization(__int64* n, __int64* p, __int64* q) {


	__int64 local_n = *n;
	__int64 i;
	__int64 idx = blockIdx.x * blockDim.x + threadIdx.x;
	__int64 step_size = blockDim.x*gridDim.x;
	__int64 steps = local_n/2;
	//TODO: n = sqrt(n) !

	for (i=3+idx; i<steps; i += step_size)
	{
		if(local_n % i == 0)
		{
			*p = i;
		}
	}
	*q = local_n / *p;

	__syncthreads();
}
